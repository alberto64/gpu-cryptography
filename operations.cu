
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

/**
* addCuda: A method that add two arrays and places the result in a third array using 
* multithreading for index calculation.
*/
__global__ void addCUDA(const int *threadCountList, const int *randNumList, int *resultList) { 
	int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = threadCountList[idx] + randNumList[idx]; 
}

/**
* subCuda: A method that substract two arrays and places the result in a third array using 
* multithreading for index calculation.
*/
__global__ void subCUDA(const int *threadCountList, const int *randNumList, int *resultList) { 
	int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = threadCountList[idx] - randNumList[idx]; 
}

/**
* multCuda: A method that multiplies two arrays and places the result in a third array using 
* multithreading for index calculation.
*/
__global__ void multCUDA(const int *threadCountList, const int *randNumList, int *resultList) { 
	int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = threadCountList[idx] * randNumList[idx]; 
}

/**
* modCuda: A method that does the modulus between two arrays and places the result in a third 
* array using multithreading for index calculation.
*/
__global__ void modCUDA(const int *threadCountList, const int *randNumList, int *resultList) { 
	int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = threadCountList[idx] % randNumList[idx]; 
}

/**
* printArray: A method that takes in an a label and an array with its size and it feeds it to printf.
*/
void printArray(const char* name, int *array, int size) {
	printf("\n%s: [ ", name);
	for(int idx = 0; idx < size; idx++) {
		printf("%i ", array[idx]);
	}
	printf("]\n");
}

/**
* runOperations: Taking the number of blocks and threads it does 4 operations on the two 
* given arrays and prints their results. Uses paged memory
*/
void runOperations(int numBlocks, int totalThreads, int* threadCountList, int* randNumList) { 
	
	// Prepare result array variables
	int* addresultList = (int*) malloc(totalThreads * sizeof(int));
	int* subresultList = (int*) malloc(totalThreads * sizeof(int));
	int* multresultList = (int*) malloc(totalThreads * sizeof(int));
	int* modresultList = (int*) malloc(totalThreads * sizeof(int));
	
	// Prepare cuda variables
	int* dev_threadCountList, *dev_randNumList, *dev_resultList;
	hipMalloc((void**)&dev_threadCountList, totalThreads * sizeof(int));
	hipMalloc((void**)&dev_randNumList, totalThreads * sizeof(int));
	hipMalloc((void**)&dev_resultList, totalThreads * sizeof(int));

	// Copy inputs into device memory 
	hipMemcpy(dev_threadCountList, threadCountList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_randNumList, randNumList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	
	// Execute each operation and bring result from device to host
	addCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	hipMemcpy(addresultList, dev_resultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 

	subCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	hipMemcpy(subresultList, dev_resultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 

	multCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	hipMemcpy(multresultList, dev_resultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 

	modCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	hipMemcpy(modresultList, dev_resultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 

	// Turned of to minimize printing
	// printArray("Add Result", addresultList, totalThreads);
	// printArray("Sub Result", subresultList, totalThreads);
	// printArray("Mult Result", multresultList, totalThreads);
	// printArray("Mod Result", modresultList, totalThreads);
	
	// Free reserved memory
	hipFree(dev_threadCountList);
	hipFree(dev_randNumList);
	hipFree(dev_resultList);
}

/**
* runOperationsOnHost: Taking the number of blocks and threads it does 4 operations on the two 
* given arrays and prints their results. Uses pinned memory
*/
void runOperationsOnHost(int numBlocks, int totalThreads, int* threadCountList, int* randNumList) { 
	
	// Prepare result array variables
	int* addresultList = (int*) malloc(totalThreads * sizeof(int));
	int* subresultList = (int*) malloc(totalThreads * sizeof(int));
	int* multresultList = (int*) malloc(totalThreads * sizeof(int));
	int* modresultList = (int*) malloc(totalThreads * sizeof(int));
	hipHostMalloc((void**)&addresultList, totalThreads * sizeof(int), hipHostMallocDefault);
	hipHostMalloc((void**)&subresultList, totalThreads * sizeof(int), hipHostMallocDefault);
	hipHostMalloc((void**)&multresultList, totalThreads * sizeof(int), hipHostMallocDefault);
	hipHostMalloc((void**)&modresultList, totalThreads * sizeof(int), hipHostMallocDefault);

	// Prepare cuda variables
	int* dev_threadCountList, *dev_randNumList, *dev_addresultList, *dev_subresultList, *dev_multresultList, *dev_modresultList;

	// Copy inputs into device memory
	hipHostGetDevicePointer(&dev_threadCountList, threadCountList, 0);
	hipHostGetDevicePointer(&dev_randNumList, randNumList, 0);
	
	// Execute each operation and bring result from device to host
	hipHostGetDevicePointer(&dev_addresultList, addresultList, 0);
	addCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_addresultList);

	hipHostGetDevicePointer(&dev_subresultList, subresultList, 0);
	subCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_subresultList);

	hipHostGetDevicePointer(&dev_multresultList, multresultList, 0);
	multCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_multresultList);

	hipHostGetDevicePointer(&dev_modresultList, modresultList, 0);
	modCUDA<<<numBlocks,totalThreads>>> (dev_threadCountList, dev_randNumList, dev_modresultList);

	// Synchonize data between device and host
	hipDeviceSynchronize();

	// Turned of to minimize printing
	// printArray("Add Result", addresultList, totalThreads);
	// printArray("Sub Result", subresultList, totalThreads);
	// printArray("Mult Result", multresultList, totalThreads);
	// printArray("Mod Result", modresultList, totalThreads);
	
	// Free reserved memory
	hipFree(dev_threadCountList);
	hipFree(dev_randNumList);
	hipFree(dev_addresultList);
	hipFree(dev_subresultList);
	hipFree(dev_multresultList);
	hipFree(dev_modresultList);
	hipHostFree(addresultList);
	hipHostFree(subresultList);
	hipHostFree(multresultList);
	hipHostFree(modresultList);
}

int main(int argc, char** argv)
{
	// Based on the work of Andrew Krepps
	
	// Set default values in case arguments don't come in command line.
	int totalThreads = 1024;
	int blockSize = 256;

	// read command line arguments
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	printf("Total Threads: %d\nBlock Size: %d\n", totalThreads, blockSize);

	// Set up variables for timing
	clock_t start, end;
	double timePassedMiliSeconds;

	// Set up paged memory space 
	int* threadCountList = (int*) malloc(totalThreads * sizeof(int));
	int* randNumList = (int*) malloc(totalThreads * sizeof(int));
	
	// Set up pinned memory space
	int* pinned_threadCountList;
	int* pinned_randNumList;
	hipHostMalloc((void**)&pinned_threadCountList, totalThreads * sizeof(int), hipHostMallocDefault);
	hipHostMalloc((void**)&pinned_randNumList, totalThreads * sizeof(int), hipHostMallocDefault);

	// Populate paged memory arrays
	for ( int idx = 0; idx < totalThreads; idx++ ) {
    	threadCountList[idx] = idx; 
		randNumList[idx] = rand() % 4;
   	}

  	// Populate pinned memory arrays
	memcpy(pinned_threadCountList, threadCountList, totalThreads * sizeof(int));  
	memcpy(pinned_randNumList, randNumList, totalThreads * sizeof(int));
	
	// Show generated values
	// Turned of to minimize printing
	// printArray("Thread Count List", threadCountList, totalThreads);
	// printArray("Random Number List", randNumList, totalThreads);
	
	// Run and time operations using paged memory
	start = clock();
	runOperations(numBlocks, totalThreads, threadCountList, randNumList);
	end = clock();
	timePassedMiliSeconds = (double) (end - start) * 1000 / CLOCKS_PER_SEC;
	printf("\nPaged Memory Time: %f Miliseconds\n", timePassedMiliSeconds);

	// Run and time operations using paged memory
	start = clock();
	runOperationsOnHost(numBlocks, totalThreads, pinned_threadCountList, pinned_randNumList);
	end = clock();
	timePassedMiliSeconds = (double) (end - start) * 1000 / CLOCKS_PER_SEC;
	printf("\nPinned Memory Time: %f Miliseconds\n", timePassedMiliSeconds);

	// Free reserved memory
	hipHostFree(pinned_threadCountList);
	hipHostFree(pinned_randNumList);
	
	return 0;
}
